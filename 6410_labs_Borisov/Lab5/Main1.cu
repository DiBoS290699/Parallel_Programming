
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iostream>

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
#define IDX2C(i,j,ld) (((j)*(ld))+(i))


void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator 
    hiprandGenerator_t prng; 
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT); 
    // Set the seed for the random number generator using the system clock 
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock()); 

    // Fill the array with random numbers on the device 
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); 
}

void single_thread_mmul(const float *A, const float *B, float *C, const int N) {
    int i, j, k;
    for(i=0; i < N; i++){
	for(j=0; j < N; j++){
 	    C[IDX2C(i, j, N)] = 0;
	    for(k=0; k < N; k++)
 		C[IDX2C(i, j, N)] += A[IDX2C(i, k, N)]*B[IDX2C(k, j, N)];
	}
    }
}


void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda = m;
    int ldb = k;
    int ldc = m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasHandle_t handle; 
    hipblasCreate(&handle); 
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc); 
    hipblasDestroy(handle); 
}


void print_matrix(const float *matrix, const int rows, const int cols) {
    std::cout << "[\n";
    for (int i = 0; i < rows; ++i) {
        std::cout << "[";
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[IDX2C(i, j, cols)] << ",\t";
        }
        std::cout << "],\n";
    }
    std::cout << "]\n";
}

int main(int argc, char* argv[]) { 
    int n = atoi(argv[1]); 
    std::cout << "NxN == " << n << "x" << n << "\n";
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C; 
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = n; 
    size_t malloc_size = nr_rows_A * nr_cols_A * sizeof(float);

    float *h_A = (float *)malloc(malloc_size); 
    float *h_B = (float *)malloc(malloc_size); 
    float *h_C = (float *)malloc(malloc_size); 

    float *d_A, *d_B, *d_C; 
    hipMalloc(&d_A, malloc_size); 
    hipMalloc(&d_B, malloc_size); 
    hipMalloc(&d_C, malloc_size); 

    if (n == 3) {
        for (int j = 0; j < 9; j += 3) {
	    h_A[j] = 1;
	    h_A[j+1] = 2;
	    h_A[j+2] = 3;
        }

        for (int j = 0; j < 9; j += 3) {
	    h_B[j] = 9;
	    h_B[j+1] = 8;
	    h_B[j+2] = 7;
        }

        hipMemcpy(d_A, h_A, malloc_size, hipMemcpyHostToDevice); 
        hipMemcpy(d_B, h_B, malloc_size, hipMemcpyHostToDevice); 
    }
    else {
   	GPU_fill_rand(d_A, nr_rows_A, nr_cols_A); 
   	GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
        hipMemcpy(h_A, d_A, malloc_size, hipMemcpyDeviceToHost); 
        hipMemcpy(h_B, d_B, malloc_size, hipMemcpyDeviceToHost); 
    }

    if (n <= 3) {
        std::cout << "A =" << std::endl; 
        print_matrix(h_A, nr_rows_A, nr_cols_A); 
        std::cout << "B =" << std::endl; 
        print_matrix(h_B, nr_rows_B, nr_cols_B);
    }

    hipEvent_t start, stop; 
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B); 

    //cudaDeviceSynchronize();
    hipEventRecord(stop, 0);
    
    hipMemcpy(h_C, d_C, malloc_size, hipMemcpyDeviceToHost); 

    if (n <= 3) {
        std::cout << "C =" << std::endl; 
        print_matrix(h_C, nr_rows_C, nr_cols_C); 
    }

    hipEventElapsedTime(&gpuTime, start, stop); 
    std::cout << "Time spent executing: " << gpuTime/1000 <<" seconds." << std::endl; 
 
    hipFree(d_A); 
    hipFree(d_B); 
    hipFree(d_C);   
 
    free(h_A); 
    free(h_B); 
    free(h_C); 
    return 0;
}
