
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iostream>
#include <ctime>

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
#define IDX2C(i,j,ld) (((j)*(ld))+(i))


void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator 
	hiprandGenerator_t prng; 
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT); 
	// Set the seed for the random number generator using the system clock 
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock()); 

	// Fill the array with random numbers on the device 
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A); 
}

void single_thread_mmul(const float *A, const float *B, float *C, const int N) {
	int i, j, k;
	for(i=0; i < N; i++){
		for(j=0; j < N; j++){
 			C[IDX2C(i, j, N)] = 0;
	    		for(k=0; k < N; k++)
 				    C[IDX2C(i, j, N)] += A[IDX2C(i, k, N)]*B[IDX2C(k, j, N)];
		}
	}
}


void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m;
	int ldb = k;
	int ldc = m;	
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	hipblasHandle_t handle; 
	hipblasCreate(&handle); 
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc); 
	hipblasDestroy(handle); 
}


void print_matrix(const float *matrix, const int rows, const int cols) {
	std::cout << "[\n";	
	for (int i = 0; i < rows; ++i) {
		std::cout << "[";
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[IDX2C(i, j, cols)] << ",\t";
        }
        std::cout << "],\n";
    }
    std::cout << "]\n";
}

int main(int argc, char* argv[]) { 
    int n = atoi(argv[1]); 
    std::cout << "NxN == " << n << "x" << n << "\n";
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C; 
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = n; 
    size_t malloc_size = nr_rows_A * nr_cols_A * sizeof(float);
    float *h_A, *h_B, *h_C, *h_1_thread_C;
    float AvgTimeForCUDA = 0.0f;
    float AvgTimeFor1Thr = 0.0f;
    float time_clock = 0.0f;
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    for (int iter=0; iter < 13; iter++) {
        h_A = (float *)malloc(malloc_size); 
        h_B = (float *)malloc(malloc_size); 	
        h_C = (float *)malloc(malloc_size); 
        h_1_thread_C = (float *)malloc(malloc_size); 
 	
        float *d_A, *d_B, *d_C; 
        hipMalloc(&d_A, malloc_size); 
        hipMalloc(&d_B, malloc_size); 
        hipMalloc(&d_C, malloc_size);

        if (n == 3) {
            for (int j = 0; j < 9; j += 3) {
                h_A[j] = 1;
                h_A[j+1] = 2;
                h_A[j+2] = 3;
            }

            for (int j = 0; j < 9; j += 3) {
                h_B[j] = 9;
                h_B[j+1] = 8;
                h_B[j+2] = 7;
            }

            hipMemcpy(d_A, h_A, malloc_size, hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, malloc_size, hipMemcpyHostToDevice);
        }
        else {
            GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
            GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
            hipMemcpy(h_A, d_A, malloc_size, hipMemcpyDeviceToHost);
            hipMemcpy(h_B, d_B, malloc_size, hipMemcpyDeviceToHost);
        }

        if (n <= 3) {
            std::cout << "A =" << std::endl;
                print_matrix(h_A, nr_rows_A, nr_cols_A);
                std::cout << "B =" << std::endl;
                print_matrix(h_B, nr_rows_B, nr_cols_B);
            }

        gpuTime = 0.0f;
        hipEventRecord(start, 0);

        gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipMemcpy(h_C, d_C, malloc_size, hipMemcpyDeviceToHost);
        hipEventElapsedTime(&gpuTime, start, stop);
        std::cout << "gpuTime = " << gpuTime / 1000 << std::endl;

        time_clock = clock();
        single_thread_mmul(h_A, h_B, h_1_thread_C, n);
        time_clock = clock() - time_clock;

        std::cout << "OneThrTime = " << time_clock / CLOCKS_PER_SEC << std::endl;
    

        if (n <= 3) {
                std::cout << "C (CUDA) =" << std::endl;
                print_matrix(h_C, nr_rows_C, nr_cols_C);
                std::cout << "C (1 thread) =" << std::endl;
                print_matrix(h_1_thread_C, nr_rows_C, nr_cols_C);
        }

        if (iter != 0) {
            AvgTimeForCUDA = AvgTimeForCUDA  + gpuTime / 1000;
            AvgTimeFor1Thr = AvgTimeFor1Thr  + time_clock / CLOCKS_PER_SEC;
        }

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        free(h_A);
        free(h_B);
        free(h_C);
        free(h_1_thread_C);

    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("AVERAGE TIME OF WORK OF CUDA IS %f \n", AvgTimeForCUDA / 12); 
    printf("AVERAGE TIME OF WORK OF 1 THREADS IS %f \n", AvgTimeFor1Thr / 12);

    return 0;
}
