#include "hip/hip_runtime.h"
__global__ void addKernel(int *c, int *a, int *b, unsigned int size)
{
	// ��� ������������� ��� ������� ����

	// ����������� ������
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// ���������� ������� c
	c[index] = a[index] + b[index];
}

#define kernel addKernel
#include "Main.h"